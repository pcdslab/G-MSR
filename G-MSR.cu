#include "hip/hip_runtime.h"
/*
This program accepts a folder of .dta files and percentage data 
to be retained. Output is generated in the form of a .ms2 file


Copyright (C) Muaaz Gul Awan and Fahad Saeed 
This program is free software; you can redistribute it and/or
modify it under the terms of the GNU General Public License
as published by the Free Software Foundation; either version 2
of the License, or (at your option) any later version.
This program is distributed in the hope that it will be useful,
but WITHOUT ANY WARRANTY; without even the implied warranty of
MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
GNU General Public License for more details.
You should have received a copy of the GNU General Public License
along with this program; if not, write to the Free Software
Foundation, Inc., 51 Franklin Street, Fifth Floor, Boston, MA  02110-1301, USA.
*/



#include <iostream>
#include <stdlib.h>
#include <math.h>
#include <cmath>
#include <stdio.h>
#include <fstream>
#include <dirent.h>
#include <vector>
#include <sstream>
#include <algorithm>
#include <unistd.h>
#include <string>
#include "specFile.h"
#include <thrust/device_ptr.h>
#include <thrust/device_vector.h>
#include <thrust/host_vector.h>
#include <thrust/sort.h>
#include <thrust/reduce.h>
#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>
#include <utility>
#include <thrust/scan.h>
#include <time.h>
#include <ctime>
#include <iomanip> 
#include<random>
#include <thrust/execution_policy.h>
#include <fstream>
#define elesPerBucket 20
#define sampleRateSo 10

using namespace std;

template <typename mType>
struct dataArrays{
	mType* dataList;
	int *prefixArray;
};

spectrum stringToSpectrum(string input);

float* headLineTokenizer(string line);

int* fileNameTokenizer(string line);

void ms2FileWriter (vector< specFile> listOfFiles, int *binSpecs, vector< dataArrays<int> >  DataVec, int arrayPerPass, int passes, int maxSize);

vector<string> listFilesOfFolder (string dir);

template <class type> 
__device__ void  swapD (type &a, type &b);

template <class type> 

__device__ void insertionSort(type *input, int begin, int end);

__device__ void mergesort (int beg, int *a, int *b, long num);

__device__ void getMinMax(int input[], int beginPtr, int endPtr, int *ret);
__global__ void getNumOfBuckets(int *prefixSumArray, int *numOfBucketsArray, int offset, int arraysPerPass);

template <typename mType>
__device__ void getSplitters (mType *data, mType *splittersArray, int sample[], int beginPtr, int endPtr, int arraySize, int *prefixBucketsArray, int offset);

template <typename mType>
__global__ void splitterKer(mType *data, mType *splittersArray, int *prefixSizeArray, int *prefixBucketsArray, int offset, int arraysPerPass);

template <typename mType>
__device__ void getBuckets(mType *input, mType *splitters, int beginPtr, int endPtr, int *bucketsSize, int *prefixBucketsArray, int offset, int maxSize, int maxBucks, int arraysPerPass);

template <typename mType>
__global__ void bucketKernel(mType *data, mType *splittersArray, int *prefixSizeArray, int *prefixBucketsArray, int *bucketSizes, int offset, int maxSize, int maxBucks, int arraysPerPass);

template <typename mType>	
__global__ void sortBuckets(mType *buckets, int *bucketsSize, int *prefixBucketsArray, int *prefixSizeArray, int offset, int arraysPerPass);

__global__ void rangeKernel(int *intens, int *scanSizes, unsigned long int *d_localRanges, int *maxAvgs, int arraysPerPass);

__device__ void quantization(int *localSpec, int size, int quanta, int *quantizedSpec, int *maxAvgs, int *quantaSizes, int *quantaBegPtr);

__device__ bool notInArray(int num,int *arr, int size);

__device__ void randomSample(int peaksReq, int *randsArray, int max);

__device__ void sampling(int *sampledSpec, int *quantizedSpec, int size, int *quantaSizes, int *quantaBegPtr, int peaksReq, int quanta, int *sampledSpecSize);

__global__ void classKernel(int *intens, int *sizesScanned, unsigned long int *localRanges, int *globalRange, int *maxAvgs, int *sampledSpecSize, int *d_sampleRate, int arraysPerPass);

template <typename mType>		
int* gpuArraySort(vector< dataArrays<mType> > newData, int numOfStreams, long long int totArrays, int maxSize, int *h_sampleRate);

vector<specFile> retrieveSpectraFromFiles(vector<string> listOfFiles, string dir, int start, int end, int countX);

int getTotalPeaks(vector<specFile> listOfFiles);

int getMaxSpecSize(vector<specFile> listofSpecFiles);


 

int main (int argc, char* argv[])
{
 

 string dirn = argv[1];
 //int *d_sampleRate;
 int *sampleRate =new int[1];
 int timesX = stoi(argv[3]);
  *sampleRate = stoi(argv[2]);
  vector<string> listOfFileNames = listFilesOfFolder(dirn);
  vector<specFile> listofSpecFiles = retrieveSpectraFromFiles(listOfFileNames, dirn, 0, listOfFileNames.size(),timesX);
 int totalPeaks = getTotalPeaks(listofSpecFiles);
  int *h_testPtr = new int[totalPeaks];
  int *h_resultSizes = new int [listofSpecFiles.size()];
  int *h_globalRange = new int[1];
int maxSpecSize = getMaxSpecSize(listofSpecFiles);
 
  thrust::device_vector<int> d_intensity;
  thrust::device_vector<int> d_intensityNew;
  thrust::host_vector<int> h_keys(totalPeaks);
  thrust::device_vector<int> d_keys;
  int *d_sizes = new int[listofSpecFiles.size()];
 // long int index = 0;
  int *after = new int[listOfFileNames.size()];
  int *before = new int[listOfFileNames.size()];
  size_t f, t, size_heap, size_stack;

  float *header = headLineTokenizer(listofSpecFiles.at(0).headerLine);
  int *namer = fileNameTokenizer(listofSpecFiles.at(0).fileName);
  
hipSetDevice(0);
hipMemGetInfo(&f,&t);
hipDeviceSetLimit(hipLimitStackSize, 102400);
hipDeviceGetLimit(&size_heap, hipLimitMallocHeapSize);
hipDeviceGetLimit(&size_stack, hipLimitStackSize);

  

 long long int totArrays = listofSpecFiles.size();
 //long int estMemReq = (5*totArrays*sizeof(unsigned long int)+ totArrays*maxSpecSize*sizeof(int));
	hipDeviceProp_t prop;
    hipGetDeviceProperties(&prop, 0);
//	cout <<"memReq:"<<estMemReq<<endl;
	//long int GPUMem = prop.totalGlobalMem*0.6;
	int estPasses = 4;//ceil((double)estMemReq/(GPUMem));
	int arraysPerPass = totArrays/estPasses;
	int arraysLastPass = arraysPerPass + totArrays%estPasses;
	
   vector< dataArrays<int> > DataVec;
   //int sizesTemp[4];
for(int p = 0; p < estPasses-1; p++)
{
   dataArrays<int> data;//data.prefixArray = 
   hipHostMalloc((void**) &data.prefixArray, (1+arraysPerPass)*sizeof(int));//new int[numOfArrays+1]; //exclusive prefix scan
   hipHostMalloc((void**) &data.dataList, (arraysPerPass*maxSpecSize)*sizeof(int));
   
  int prefixSum = 0, q = 0, preIndex=0;
  for(int i = p*arraysPerPass; i < (p+1)*arraysPerPass; i++){
      data.prefixArray[preIndex] = prefixSum;
	  for(int j = 0; j < listofSpecFiles.at(i).listVals.size(); j++){
         data.dataList[q] = listofSpecFiles.at(i).listVals.at(j).intensity;
      
		q++;
      }
	  prefixSum += listofSpecFiles.at(i).listVals.size();
	  preIndex++;
  }
   data.prefixArray[preIndex] = prefixSum;
   DataVec.push_back(data);
   
}

//for the last pass
dataArrays<int> data;//data.prefixArray = 
   hipHostMalloc((void**) &data.prefixArray, (1+arraysPerPass)*sizeof(int));//new int[numOfArrays+1]; //exclusive prefix scan
   hipHostMalloc((void**) &data.dataList, (arraysPerPass*maxSpecSize)*sizeof(int));
   
  int prefixSum = 0, q = 0, preIndex=0;
  for(int i = (estPasses-1)*arraysPerPass; i < (estPasses-1)*arraysPerPass + arraysLastPass; i++){
      data.prefixArray[preIndex] = prefixSum;
	  for(int j = 0; j < listofSpecFiles.at(i).listVals.size(); j++){
         data.dataList[q] = listofSpecFiles.at(i).listVals.at(j).intensity;
        // h_keys[index] = i;
        // index++;
		q++;
      }
	  prefixSum += listofSpecFiles.at(i).listVals.size();
	  preIndex++;
  }
  //sizesTemp[3] = q;
   data.prefixArray[preIndex] = prefixSum;
   DataVec.push_back(data);

	for(int i = 0; i < estPasses-1; i++){
		//cout <<"for Sizes:"<<sizesTemp[i]<<"prefix:"<<DataVec.at(i).prefixArray[arraysPerPass]<<endl;
		
		}
	//	cout <<"for Sizes:"<<sizesTemp[3]<<"prefix:"<<DataVec.at(3).prefixArray[arraysLastPass]<<endl;
		
  // cout<<"total spectra::::::"<<totArrays<<endl;
   
  //  cout <<"*********file reading done calling sort**********"<<endl;
   int *h_result = gpuArraySort<int>(DataVec, 1, listofSpecFiles.size(), maxSpecSize, sampleRate );
	
  ///writing takes time so for now not writing.
	//  ms2FileWriter (listofSpecFiles, h_result, DataVec[0].prefixArray);
    ms2FileWriter (listofSpecFiles, h_result, DataVec, arraysPerPass, estPasses, maxSpecSize);

  
 

 // hipFree(d_localRanges);
 // hipFree(d_maxAvgs);
 // hipFree(d_globalRange);
  //hipFree(d_sampledSpecSizes);

  return 0;

}







//converts string to spectrum class
spectrum stringToSpectrum(string input) {
	spectrum newSpectrum;
	int point = 0;
	point = input.find(" ");
	newSpectrum.m_z = stof(input.substr(0, point));
	newSpectrum.intensity = stof(input.substr(point + 1, input.find('\0')));


	return newSpectrum;
}

//tokenizes the header line 
float* headLineTokenizer(string line)
{
	float *headerFile = new float[2];
	
	headerFile[0] = stof(line.substr(0, line.find(" ")));
	headerFile[1] = stof(line.substr(line.find(" ")+1, line.find('\0')));
	
	return headerFile;
}

//tokenizes the file name
int* fileNameTokenizer(string line)
{
	int *fileNameTokens = new int[1];
	
	fileNameTokens[0] = stoi(line.substr(line.find(".")+1, line.find(".5", line.find(".")+1, 1)-1));
	
	return fileNameTokens;
}

//writes to .ms2 file
void ms2FileWriter (vector< specFile> listOfFiles, int *binSpecs, vector< dataArrays<int> >  DataVec, int arrayPerPass, int passes, int maxSize)
{
	
	ofstream myFileWriter;
	string targetPath = "reducedSpec.ms2";
	myFileWriter.open(targetPath.c_str());
	for(int p = 0; p < passes; p++)
	{
	for(int i = 0; i < arrayPerPass; i++)
	{
		float *header = headLineTokenizer(listOfFiles.at(arrayPerPass*p + i).headerLine);
		int *namer = fileNameTokenizer(listOfFiles.at(arrayPerPass*p + i).fileName);
		float prec_mass_peptide = (float) ((header[0]) + (header[1] - 1)) / (header[1]);
		int specStart = DataVec.at(p).prefixArray[i];
		int specEnd = DataVec.at(p).prefixArray[i+1];
		int specSize = specEnd - specStart;
		myFileWriter << setprecision(8);
		myFileWriter << "S\t"<<namer[0]<<"\t"<<namer[0]<<"\t"<<prec_mass_peptide<<endl;
		myFileWriter << "Z\t"<<header[1]<<"\t"<<header[0]<<endl;
				
			
		for(int j = 0; j < specSize; j++)
		{
			if(binSpecs[p*arrayPerPass*maxSize + specStart + j] == 1)
			{
				myFileWriter << listOfFiles.at(arrayPerPass*p + i).listVals.at(j).m_z<<"\t"<<listOfFiles.at(arrayPerPass*p + i).listVals.at(j).intensity<<endl;
			}
		}
		delete[] header;
		delete[] namer;
	}
	}
}

//lists file for folder
vector<string> listFilesOfFolder (string dir)
{
	
	vector<string> listOfFiles;
    DIR *dp;
    struct dirent *dirp;
    if((dp  = opendir(dir.c_str())) == NULL) {
        cout << "Error opening the file " << dir << endl;
    }

    while ((dirp = readdir(dp)) != NULL) {
        if(string(dirp->d_name).compare("..") == 0 || string(dirp->d_name).compare(".") == 0)
			continue;
		else
        listOfFiles.push_back(string(dirp->d_name));
    }
    closedir(dp);
    return listOfFiles;
}



//swap function for Insertion sort
template <class type> __device__ void  swapD (type &a, type &b)

{
    type temp;
        temp=a;
	a=b;
        b=temp;
}

//insertion sort
template <class type> __device__ void insertionSort(type *input, int begin, int end){
	int i, j; 
	for (i = begin+1; i < end; i++) {
		j = i;
	while (j > begin && input[j - 1] > input[j]) {
		swapD(input[j], input[j-1]);
		j--;
		}
	}
}


// mergesort code from stackoverflow
__device__ void mergesort (int beg, int *a, int *b, long num)
{
    int rght, rend;
    int i,j,m;

    for (int k=1; k < num; k *= 2 ) {       
        for (int left=beg+0; left+k < num; left += k*2 ) {
            rght = left + k;        
            rend = rght + k;
            if (rend > num) rend = num; 
            m = left; i = left; j = rght; 
            while (i < rght && j < rend) { 
                if (a[i] <= a[j]) {         
                    b[m] = a[i]; i++;
                } else {
                    b[m] = a[j]; j++;
                }
                m++;
            }
            while (i < rght) { 
                b[m]=a[i]; 
                i++; m++;
            }
            while (j < rend) { 
                b[m]=a[j]; 
                j++; m++;
            }
            for (m=left; m < rend; m++) { 
                a[m] = b[m]; 
            }
        }
    }
}




__device__ void getMinMax(int input[], int beginPtr, int endPtr, int *ret){
          int min = input[beginPtr];
          int max = 0;
          for(int i = beginPtr; i < endPtr; i++){
              if(min > input[i])
                  min = input[i];
              if (max < input[i])
                  max = input[i];     
            }

     ret[0] = min;
     ret[1] = max;

}


//kernel for obtaining num of buckets for each array
__global__ void getNumOfBuckets(int *prefixSumArray, int *numOfBucketsArray, int offset, int arraysPerPass){
	int id = offset + blockIdx.x; 
	
	if(id < arraysPerPass)
		numOfBucketsArray[id] = (prefixSumArray[id+1] - prefixSumArray[id])/elesPerBucket;
}

template <typename mType>
__device__ void getSplitters (mType *data, mType *splittersArray, int sample[], int beginPtr, int endPtr, int arraySize, int *prefixBucketsArray, int offset){
           int SAMPLED = (sampleRateSo*arraySize)/100;           
		   mType *mySamples = &sample[SAMPLED];
		   mType *mySamples_temp = &mySamples[SAMPLED];
		   int bid = blockIdx.x + offset;
		   
			// calculating samples for this array
			int numOfSamples = ((float)sampleRateSo/100)*(arraySize);
			//calculating the number of buckets for this array
			int numOfBuckets = (bid == 0) ? prefixBucketsArray[0] : (prefixBucketsArray[bid] - prefixBucketsArray[bid-1]);
			
            for(int i = 0; i < numOfSamples; i++)
	           mySamples[i] = data[beginPtr+sample[i]];

	         mergesort (0, mySamples, mySamples_temp, numOfSamples);
	        //calculate splitter index for this array 
            int splitterIndex = ((bid == 0)? 1 : (prefixBucketsArray[bid-1]+1)+1); //the other plus one is for leaving space for smallest splitter(added later)
            int splittersSize=0;
	        for(int i = (numOfSamples)/(numOfBuckets); splittersSize < numOfBuckets-1; i +=(numOfSamples)/(numOfBuckets)){
                 splittersArray[splitterIndex] = mySamples[i];
                 splitterIndex++;
                 splittersSize++;
             }
			int bits = 8*sizeof(mType);
            mType min = -(1 << (bits-1));
            mType max = (1 << (bits - 1)) - 1;
            splittersArray[((bid == 0)? 0 : (prefixBucketsArray[bid-1]+1))] = min;//smaller than min value
            splittersArray[((bid == 0)? prefixBucketsArray[0] : (prefixBucketsArray[bid]))] = max;//larger than max value;
      
}

//kernel for obtaining splitters
template <typename mType>
__global__ void splitterKer(mType *data, mType *splittersArray, int *prefixSizeArray, int *prefixBucketsArray, int offset, int arraysPerPass){
          int bid = blockIdx.x + offset;
		  if(bid < arraysPerPass){
             int id = offset + blockIdx.x;
	         extern __shared__ int majorArray[];
	         int *sampleSh = majorArray;
			 int arraySize = prefixSizeArray[id+1] - prefixSizeArray[id];
			// calculating samples for this array
			int numOfSamples = ((float)sampleRateSo/100)*(arraySize);
            int max = arraySize;
            int  sam = numOfSamples;
            int stride = max/sam;
	        int sampleVal = 0;
            for( int i = 0; i < numOfSamples; i++)
            {
               sampleSh[i] = sampleVal;
               sampleVal += stride; 
            }
			 
	        getSplitters(data, splittersArray, sampleSh, prefixSizeArray[id], prefixSizeArray[id+1], prefixSizeArray[id+1] - prefixSizeArray[id], prefixBucketsArray, offset);

           }
        }

		
template <typename mType>
__device__ void getBuckets(mType *input, mType *splitters, int beginPtr, int endPtr, int *bucketsSize, int *prefixBucketsArray, int offset, int maxSize, int maxBucks, int arraysPerPass){
     int bid = blockIdx.x + offset;
	 int numOfBuckets = (bid == 0) ? prefixBucketsArray[0] : (prefixBucketsArray[bid] - prefixBucketsArray[bid-1]);
        
	if(blockIdx.x < arraysPerPass && threadIdx.x < numOfBuckets){
	  int *localSizes = &splitters[maxBucks+2];
	  int id = threadIdx.x;
	  int sizeOffset = (bid == 0) ? (0+threadIdx.x) : (prefixBucketsArray[bid-1] + threadIdx.x);  //blockIdx.x*BUCKETS+threadIdx.x;
	 // int sizeOffsetBlock = (bid == 0) ? (0) : (prefixBucketsArray[bid-1]);
      int bucketSizeOff = sizeOffset+1;
	  //__shared__ int my
      mType myBucket[750]; 
      int indexSum=0;
      localSizes[threadIdx.x] = 0;
	  
     for(int i = beginPtr; i < endPtr ; i++)
	 {
         if(input[i] > splitters[id] && input[i] <= splitters[id+1])
		 {
			myBucket[localSizes[threadIdx.x]] = input[i];
			localSizes[threadIdx.x]++;
         }
     }
       
    __syncthreads();
         //prefix sum for bucket sizes of current array
     for(int j = 0; j < threadIdx.x; j++)
        indexSum += localSizes[j];

     bucketsSize[bucketSizeOff] = localSizes[threadIdx.x];
         //writing back current buckt back to the input memory
	 for(int i = 0; i < localSizes[threadIdx.x]; i++)
             input[indexSum+beginPtr+i] = myBucket[i];
		
	}
     

}
		
//kernel for obtaining buckets
template <typename mType>
__global__ void bucketKernel(mType *data, mType *splittersArray, int *prefixSizeArray, int *prefixBucketsArray, int *bucketSizes, int offset, int maxSize, int maxBucks, int arraysPerPass){
    
	int numOfBuckets = (offset + blockIdx.x == 0) ? prefixBucketsArray[0] : (prefixBucketsArray[offset + blockIdx.x] - prefixBucketsArray[(blockIdx.x + offset)-1]);
        
	if(blockIdx.x < arraysPerPass && threadIdx.x < numOfBuckets){
        bucketSizes[0] = 0;
		extern __shared__ int majorArray[];
		int bid = offset + blockIdx.x;
        int arrBegin = prefixSizeArray[bid];
        int arrEnd = prefixSizeArray[bid+1];
		    
        int splitterIndexSt = ((bid == 0)? 0 : (prefixBucketsArray[bid-1]+1));//blockIdx.x*(BUCKETS+1);
        mType *splitters = majorArray;
 
        //int j = 0;
		
	   	if(threadIdx.x == numOfBuckets - 1)
			{
             splitters[threadIdx.x] = splittersArray[splitterIndexSt+threadIdx.x];
             splitters[threadIdx.x+1] = splittersArray[splitterIndexSt+threadIdx.x+1];
		    } 	
            else	
           	 splitters[threadIdx.x] = splittersArray[splitterIndexSt+threadIdx.x];	
		
	   
		 __syncthreads();
	    getBuckets(data, splitters, arrBegin, arrEnd, bucketSizes, prefixBucketsArray, offset, maxSize, maxBucks, arraysPerPass);

	}
}		
		
	
		
		
		
//sorting kernel	
template <typename mType>	
__global__ void sortBuckets(mType *buckets, int *bucketsSize, int *prefixBucketsArray, int *prefixSizeArray, int offset, int arraysPerPass){
	int bid = blockIdx.x + offset;
	int numOfBuckets = (bid == 0) ? prefixBucketsArray[0] : (prefixBucketsArray[bid] - prefixBucketsArray[bid-1]);
     
	
       if(bid < arraysPerPass && threadIdx.x < numOfBuckets){
		int sizeOffset = (bid == 0) ? (0+threadIdx.x) : (prefixBucketsArray[bid-1] + threadIdx.x); 
        int sizeOffsetBlock = (bid == 0) ? (0) : (prefixBucketsArray[bid-1]);
       
      //  int tid = threadIdx.x;
		//int arraySize = prefixSizeArray[bid+1] - prefixSizeArray[bid];
	   
        int indexSum = 0;
    

        //prefix sum for bucket sizes of current array
        
     	  for(int j = 0; j < threadIdx.x; j++)
              indexSum += bucketsSize[sizeOffsetBlock+j+1];

 
          insertionSort(buckets, prefixSizeArray[bid] + indexSum,prefixSizeArray[bid] + indexSum + bucketsSize[sizeOffset+1]);
		
   		__syncthreads();
 
}


}

__global__ void rangeKernel(int *intens, int *scanSizes, unsigned long int *d_localRanges, int *maxAvgs, int arraysPerPass){
       int totalSpecs = arraysPerPass;
       int prefixSum = 0;
       int min3[10];
       int max3[10];
       int localRange =0, minAvg=0, maxAvg=0;
       int size = 0;
   
          size = scanSizes[blockIdx.x+1] - scanSizes[blockIdx.x];
          prefixSum = scanSizes[blockIdx.x];

       

       
       if(blockIdx.x < totalSpecs){
       
       for(int i = prefixSum, index=0; i < prefixSum+10;i++,index++)
            min3[index] = intens[i]; 
       for(int i = prefixSum+(size-1), index=0; i > (prefixSum+size)-11;i--,index++)
            max3[index] = intens[i]; 
       for(int i = 0; i < 10; i++){
           minAvg += min3[i];
           maxAvg += max3[i];
        }
        
      minAvg = minAvg/10;
      maxAvg = maxAvg/10;

      maxAvgs[blockIdx.x] = maxAvg;
      localRange = maxAvg - minAvg;
      d_localRanges[blockIdx.x] = localRange; 
       }
}


__device__ void quantization(int *localSpec, int size, int quanta, int *quantizedSpec, int *maxAvgs, int *quantaSizes, int *quantaBegPtr){
           float jumpInc = (float) 1/quanta, jump = 0, jumpLag = 0;
           float refVal = maxAvgs[blockIdx.x];
           int indexPtr = 0;
           int quantumSize = 0;
           for(int i = 0; i < quanta; i++){ 
                quantumSize = 0;
                if(i == 0){
               jumpLag = 0;
               jump = jumpInc;
                }

               for(int j = 0; j < size; j++){
                   if( i == quanta -1){
               // we add only the indexes of the intensities in the final array
                      if(localSpec[j] >= (refVal*jumpLag)){
                         quantizedSpec[indexPtr] = j;
                         indexPtr++;
                         quantumSize++;
                       }
                      else{}
           
                    }
                   else{
                      if(localSpec[j] >= (refVal*jumpLag) && (localSpec[j]< (refVal*jump))){
                         quantizedSpec[indexPtr] = j;
                         indexPtr++;   
                         quantumSize++;  
                      }
                   }
                }
                quantaSizes[i] = quantumSize;
                quantaBegPtr[i] = indexPtr-quantumSize; 
                jump = jump + jumpInc;
                jumpLag += jumpInc;


           }



    }

__device__ bool notInArray(int num,int *arr, int size){
                bool var = true;
                for(int i = 0; i < size; i++){
                  if(arr[i] == num)
                     var = false;
               }
       return var;

    }

// generate random samples on device
__device__ void randomSample(int peaksReq, int *randsArray, int max){
             hiprandState_t state;
             hiprand_init(1234, blockIdx.x, 0, &state);
             int count = 0;
                        
             while(count < peaksReq){
                float randF = hiprand_uniform(&(state));
                randF *= ((max-1) - 0 + 0.999999);
                randF += 0;
                int randI = (int)truncf(randF);
                if( notInArray(randI, randsArray, count)){
                    randsArray[count] = randI;
                    count++;
                   }
                 }

}

__device__ void sampling(int *sampledSpec, int *quantizedSpec, int size, int *quantaSizes, int *quantaBegPtr, int peaksReq, int quanta, int *sampledSpecSize){
                           
        int tolPeaks = 5;
        int *rands = new int[peaksReq];
        if( (quantaSizes[quanta-1] >= (peaksReq - tolPeaks)  && (quantaSizes[quanta-1] <= (peaksReq+tolPeaks)))){
           for(int i = quantaBegPtr[quanta-1]; i <(quantaBegPtr[quanta-1]+ quantaSizes[quanta-1]); i++){
            sampledSpec[quantizedSpec[i]] = 1;
         
           }
          }

         else if( (quantaSizes[quanta-1] > (peaksReq + tolPeaks))){
              //generate random nums array
            int max = quantaSizes[quanta-1];
               randomSample (peaksReq, rands,max);
               
               for(int i = 0; i < peaksReq; i++){
                 sampledSpec[quantizedSpec[quantaBegPtr[quanta-1]+rands[i]]] = 1;
                
               }
         
          }

         else{
              while((peaksReq > quantaSizes[quanta-1]) || ((quantaSizes[quanta-1] >= (peaksReq - tolPeaks)) && (quantaSizes[quanta-1] <= (peaksReq + tolPeaks)))){
                for(int i = quantaBegPtr[quanta-1]; i < (quantaBegPtr[quanta-1]+quantaSizes[quanta-1]);i++){
                     sampledSpec[quantizedSpec[i]] = 1;
                  

                   }
                 peaksReq = peaksReq - quantaSizes[quanta-1];
                 quanta--;
                }
             int max = quantaSizes[quanta-1];
                randomSample (peaksReq, rands,max);
                
                for(int i = 0; i < peaksReq; i++){
                 sampledSpec[quantizedSpec[quantaBegPtr[quanta-1]+rands[i]]] = 1;
                 
              }

           }
    delete[] rands;
   }

    

    


__global__ void classKernel(int *intens, int *sizesScanned, unsigned long int *localRanges, int *globalRange, int *maxAvgs, int *sampledSpecSize, int *d_sampleRate, int arraysPerPass){
        int totalSpecs = arraysPerPass;
        int sample_rate = *d_sampleRate;
        int size = 0;
        int prefixSum =0;
       
           size = sizesScanned[blockIdx.x+1]-sizesScanned[blockIdx.x];
           prefixSum = sizesScanned[blockIdx.x];

        
        if( blockIdx.x < totalSpecs){
          float relativeRange = ((float)localRanges[blockIdx.x]/(*globalRange))*100;
          int peaksReq = ((float)sample_rate/100)*size;
          int *localSpec = new int[size];
          int *quantizedSpec = new int[size]; 
          int *sampledSpec = new int[size];
          int *quantaSizes = new int[11];
          int *quantaBegPtr = new int[11];

          for(int i = 0; i < size; i++)
             sampledSpec[i] = 0;       

          for(int i = 0; i < size; i++)
             localSpec[i] = intens[prefixSum+i];
        

           if(relativeRange < 25){
              quantization (localSpec, size, 5, quantizedSpec, maxAvgs, quantaSizes,quantaBegPtr);
              sampling (sampledSpec, quantizedSpec, size, quantaSizes,quantaBegPtr, peaksReq, 5, sampledSpecSize);
            }
            else if ((relativeRange >= 25) && (relativeRange < 50)){
              quantization (localSpec, size, 7, quantizedSpec, maxAvgs, quantaSizes, quantaBegPtr);
             sampling (sampledSpec, quantizedSpec, size, quantaSizes,quantaBegPtr, peaksReq, 7, sampledSpecSize);

            }  
            
            else if ((relativeRange >= 50) && (relativeRange < 75)){
              quantization (localSpec, size, 9, quantizedSpec, maxAvgs, quantaSizes, quantaBegPtr);
             sampling (sampledSpec, quantizedSpec, size, quantaSizes,quantaBegPtr, peaksReq, 9, sampledSpecSize);

            }   
            
            else if (relativeRange >= 75){
              quantization (localSpec, size, 11, quantizedSpec, maxAvgs, quantaSizes, quantaBegPtr);
             sampling (sampledSpec, quantizedSpec, size, quantaSizes,quantaBegPtr, peaksReq, 11, sampledSpecSize);

            }

       // int indexPtr = 0;

		for(int i = 0; i < size; i++)
		{
			intens[prefixSum+i] = sampledSpec[i];
		}
	

      
   
       delete[] localSpec;
       delete[] quantizedSpec;
       delete[] quantaSizes;
       delete[] quantaBegPtr;
       delete[] sampledSpec;
   }
}


template <typename mType>		
int* gpuArraySort(vector< dataArrays<mType> > newData, int numOfStreams, long long int totArrays, int maxSize, int *h_sampleRate  ){
	//long int estMemReq = (5*totArrays*sizeof(unsigned long int)+ totArrays*maxSize*sizeof(int));
	hipDeviceProp_t prop;
    hipGetDeviceProperties(&prop, 0);
//	long int GPUMem = prop.totalGlobalMem*0.6;
	int estPasses = 4;
	long long int arraysPerPass = totArrays/estPasses;
	long long int arraysPerStream = arraysPerPass/numOfStreams;

	int *h_result = new int[totArrays*maxSize];
	int *h_offsetArray = new int[numOfStreams];
	int *d_prefixSum, *d_numOfBuckets;
    mType *d_inputData, *d_splitters, *d_bucketSizes;
	mType *h_sortedData;// = new mType[newData.prefixArray[totArrays]];
	int SAMPLED = (sampleRateSo*maxSize)/100;
    int maxBuckets = (maxSize/elesPerBucket);
	 size_t size_heap, size_stack;
    //setting stack size limit
    hipDeviceSetLimit(hipLimitStackSize,20480);
    hipDeviceGetLimit(&size_heap, hipLimitMallocHeapSize);
    hipDeviceGetLimit(&size_stack, hipLimitStackSize);
   int *h_localRange, *h_maxAvgs;
	hipHostMalloc((void**)&h_sortedData, sizeof(mType)*totArrays*4*maxSize);//newData.prefixArray[totArrays]);
    hipHostMalloc((void**)&h_localRange, sizeof(int)*totArrays);//newData.prefixArray[totArrays]);
	hipHostMalloc((void**)&h_maxAvgs, sizeof(int)*totArrays);
    hipMalloc((void**) &d_prefixSum, (arraysPerPass+1)*sizeof(int));
    hipMalloc((void**) &d_numOfBuckets, (arraysPerPass+1)*sizeof(int));
    hipMalloc((void**) &d_inputData, sizeof(mType)*arraysPerPass*maxSize);//prefixSum[arraysPerPass]*
	
	hipMalloc((void**) &d_splitters, (arraysPerPass*maxBuckets+2*arraysPerPass)*sizeof(mType));
    hipMalloc((void**) &d_bucketSizes, (arraysPerPass*maxBuckets)*sizeof(int)); 
	
	
	unsigned long int *d_localRanges;
	int *d_maxAvgs, *d_globalRange, *d_sampledSpecSizes, *d_sampleRate;
	
    hipMalloc((void**) &d_localRanges, sizeof(unsigned long int)*arraysPerPass);
    hipMalloc((void**) &d_maxAvgs, sizeof(int)*arraysPerPass);
    hipMalloc((void**) &d_globalRange, sizeof(int));
	hipMalloc((void**) &d_sampleRate, sizeof(int));
	hipMalloc((void**) &d_sampledSpecSizes, sizeof(int));
	 
	hipStream_t stream[numOfStreams];
	for (int i = 0; i < numOfStreams; ++i)
          hipStreamCreate(&stream[i]) ;
	
	//creating events
	hipEvent_t start, stop, stream1, stream2, stream3, stream4, copyStart, copyStop, synch[numOfStreams];
	hipEventCreate(&start);
	hipEventCreate(&stop);
	hipEventCreate(&copyStart);
	hipEventCreate(&copyStop);
	hipEventCreate(&stream1);
	hipEventCreate(&stream2);
	hipEventCreate(&stream3);
	hipEventCreate(&stream4);
	
	for(int i = 0; i < numOfStreams; i++)
		hipEventCreate(&synch[i]);
	
   
   //  int errCout = 0;
   float copyTime = 0;
   
   thrust::device_ptr<int> prefixNumBuckets = thrust::device_pointer_cast(d_numOfBuckets);
	 
    unsigned long int *range = new unsigned long int[1];
	unsigned long int *temp_range = new unsigned long int[estPasses];
	  hipEventRecord(start); 
   for(int p = 0; p <estPasses; p++){

   //copying prefixSums to Device
   
     for(int i = 0; i < numOfStreams -1; i++)   
        hipMemcpyAsync(&d_prefixSum[i*arraysPerStream], &newData[p].prefixArray[i*arraysPerStream], sizeof(int)*(arraysPerStream), hipMemcpyHostToDevice, stream[i]);
     hipMemcpyAsync(&d_prefixSum[(numOfStreams-1)*arraysPerStream], &newData[p].prefixArray[(numOfStreams-1)*arraysPerStream],
	 sizeof(int)*(arraysPerStream+1), hipMemcpyHostToDevice, stream[numOfStreams-1]);
    
	//copying input data to device 
	
	  for(int i = 0; i < numOfStreams; i++)   
        hipMemcpyAsync(&d_inputData[newData[p].prefixArray[i*arraysPerStream]], &newData[p].dataList[newData[p].prefixArray[i*arraysPerStream]],
	    sizeof(int)*(newData[p].prefixArray[(i+1)*arraysPerStream]-newData[p].prefixArray[i*arraysPerStream]), hipMemcpyHostToDevice, stream[i]);
	
   //clculating buckets on GPU
   int offset = 0;
   for(int i = 0; i < numOfStreams; i++){
	   offset = i*arraysPerStream;
       getNumOfBuckets<<<arraysPerStream, 1, 0, stream[i]>>>(d_prefixSum, d_numOfBuckets, offset,arraysPerPass);
   }
 


   for(int i = 0; i < numOfStreams; i++){
	   if( i > 0)
	    hipStreamWaitEvent(stream[i], synch[i-1], 0);
	
   thrust::inclusive_scan(thrust::cuda::par.on(stream[i]), prefixNumBuckets+(i*arraysPerStream)+(i==0?0:-1), 
  prefixNumBuckets+(i+1)*arraysPerStream , prefixNumBuckets+(i*arraysPerStream)+(i==0?0:-1));
      hipEventRecord(synch[i],stream[i]);
	 
   }
   
	
  
   // cout<< "**** Generating Splitters ****" << endl;
      offset = 0;
    for(int i = 0; i < numOfStreams; i++){  
	   offset = i*arraysPerStream;
       splitterKer<<<arraysPerStream, 1, 3*SAMPLED*sizeof(int), stream[i]>>>(d_inputData, d_splitters, d_prefixSum, d_numOfBuckets, offset,arraysPerPass);
	   
	}
	

	//cout<< "**** Generating Buckets ****" << endl;
	
    offset = 0;
      for(int i = 0; i < numOfStreams; i++){  
	   offset = i*arraysPerStream;
       bucketKernel<<<arraysPerStream, maxBuckets, (2*maxBuckets + 2)*sizeof(int), stream[i]>>>(d_inputData, d_splitters, d_prefixSum, d_numOfBuckets, d_bucketSizes, offset, maxSize,maxBuckets,arraysPerPass);
   
	}
	

	  
	//cout<< "**** Sorting Buckets ****" << endl;
   
	offset = 0;
	   for(int i = 0; i < numOfStreams; i++){  
	   offset = i*arraysPerStream;
       sortBuckets<<<arraysPerStream, maxBuckets, 0, stream[i]>>>(d_inputData, d_bucketSizes,d_numOfBuckets, d_prefixSum, offset,arraysPerPass);
	
	}
	
		hipDeviceSynchronize();

	//hipFree(d_bucketSizes);
	//hipFree(d_splitters);
	//hipFree(d_numOfBuckets);
	
	

    rangeKernel<<<arraysPerPass,1>>>(d_inputData, d_prefixSum, d_localRanges, d_maxAvgs,arraysPerPass);
	
	
    thrust::device_ptr<unsigned long int> dPtrRange = thrust::device_pointer_cast(d_localRanges); 
  
	temp_range[p] = thrust::reduce(dPtrRange, dPtrRange+arraysPerPass);
	hipMemcpy (&h_localRange[p*arraysPerPass], d_localRanges, sizeof(unsigned long int)*arraysPerPass, hipMemcpyDeviceToHost); 
	hipMemcpy (&h_maxAvgs[p*arraysPerPass], d_maxAvgs, sizeof(int)*arraysPerPass, hipMemcpyDeviceToHost); 
  
     }// for ends
	 
	  unsigned long int number = 0;
	 
	  for(int i = 0; i < estPasses; i++)
	  {
		  number += temp_range[i];
	  }
	   number = number/totArrays;
	  *range = number;
	  
     hipMemcpy(d_globalRange, range, sizeof(int), hipMemcpyHostToDevice);
     hipMemcpy(d_sampleRate, h_sampleRate, sizeof(int), hipMemcpyHostToDevice);
	 
	 
	  //reamining part of the algorithm
	 for(int p = 0; p < estPasses; p++)
	 {
		for(int i = 0; i < numOfStreams; i++)   
        hipMemcpyAsync(&d_inputData[newData[p].prefixArray[i*arraysPerStream]], &newData[p].dataList[newData[p].prefixArray[i*arraysPerStream]],
	    sizeof(int)*(newData[p].prefixArray[(i+1)*arraysPerStream]-newData[p].prefixArray[i*arraysPerStream]), hipMemcpyHostToDevice, stream[i]);
	  
	  
	    for(int i = 0; i < numOfStreams -1; i++)   
        hipMemcpyAsync(&d_prefixSum[i*arraysPerStream], &newData[p].prefixArray[i*arraysPerStream], sizeof(int)*(arraysPerStream), hipMemcpyHostToDevice, stream[i]);
        hipMemcpyAsync(&d_prefixSum[(numOfStreams-1)*arraysPerStream], &newData[p].prefixArray[(numOfStreams-1)*arraysPerStream],
	    sizeof(int)*(arraysPerStream+1), hipMemcpyHostToDevice, stream[numOfStreams-1]);
    
	  //copying the ranges back
	    hipMemcpy(d_localRanges, &h_localRange[p*arraysPerPass], sizeof(unsigned long int)*arraysPerPass, hipMemcpyHostToDevice);
        hipMemcpy(d_maxAvgs, &h_maxAvgs[p*arraysPerPass], sizeof(int)*arraysPerPass, hipMemcpyHostToDevice);
	
	  
	    classKernel<<<arraysPerPass,1>>>(d_inputData, d_prefixSum, d_localRanges, d_globalRange, d_maxAvgs, d_sampledSpecSizes, d_sampleRate,arraysPerPass);

	  
	    for(int i = 0; i < numOfStreams; i++)   
        hipMemcpyAsync(&h_result[(newData[p].prefixArray[i*arraysPerStream]) + p*arraysPerPass*maxSize], &d_inputData[newData[p].prefixArray[i*arraysPerStream]],
	    sizeof(mType)*(newData[p].prefixArray[(i+1)*arraysPerStream]-newData[p].prefixArray[i*arraysPerStream]), hipMemcpyDeviceToHost, stream[i]);
	
		 
	 }
	 
       float milliseconds = 0;
	   hipEventRecord(stop);
	   hipEventSynchronize(stop);
	   hipEventElapsedTime(&milliseconds, start, stop);
	


	hipHostFree(newData[0].dataList);
	hipHostFree(newData[1].dataList);
	hipHostFree(newData[2].dataList);
	hipHostFree(newData[3].dataList);
	hipHostFree(h_sortedData);
	hipFree(d_prefixSum);
	
	hipFree(d_inputData);
	cout<<" time taken:"<<(milliseconds-copyTime)<<" total spectra:"<<totArrays<<endl;
	return h_result;
} 


//extracting spectra from file and converting them into specFile type.

vector<specFile> retrieveSpectraFromFiles(vector<string> listOfFiles, string dir, int start, int end, int countX) {
	vector<specFile> listofSpecFiles;
	spectrum tempSpectrum;
	string filePath;
	string line;
	int p = 0;
	for(int times = 0; times < countX; times++)
	{
		for (int i = start; i < end; i++) {
			p = 0;
			specFile tempFile;
			filePath = dir + listOfFiles.at(i);
			tempFile.fileName = listOfFiles.at(i);
			ifstream myfile(filePath.c_str());
			if (myfile.is_open())
			{
				while (getline(myfile, line))
				{
					if (p == 0) {
						p++;
						tempFile.headerLine = line;
					}
					else {
						tempSpectrum = stringToSpectrum(line);
						tempFile.listVals.push_back(tempSpectrum);
					}
				}
				myfile.close();
				listofSpecFiles.push_back(tempFile);
			}
		}
	}
	return listofSpecFiles;
}

int getTotalPeaks(vector<specFile> listOfFiles){
    int totalSize = 0;
    for(int i = 0; i < listOfFiles.size(); i++){
       totalSize = totalSize + listOfFiles.at(i).listVals.size();
    }

    return totalSize;
 }


int getMaxSpecSize(vector<specFile> listofSpecFiles)
{
	int largest = 0;
	
	for (int i = 0; i < listofSpecFiles.size(); i++)
	{
		if (largest < listofSpecFiles.at(i).listVals.size())
			largest = listofSpecFiles.at(i).listVals.size();
	}
	return largest;
}

